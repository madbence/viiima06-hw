#include <stdio.h>
#include <stdlib.h>
#include <cmath>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

struct vec3 {
  float x, y, z;
  __device__ __host__ vec3(float x = 0, float y = 0, float z = 0):x(x),y(y),z(z) {}
  __device__ __host__ vec3 operator+(const vec3& a) const { return vec3(x + a.x, y + a.y, z + a.z); }
  __device__ __host__ vec3 operator-(const vec3& a) const { return vec3(x - a.x, y - a.y, z - a.z); }
  __device__ __host__ float operator*(const vec3& a) const { return x * a.x + y * a.y + z * a.z; }
  __device__ __host__ vec3 operator%(const vec3& a) const { return vec3(y * a.z - z * a.y, x * a.z - z * a.x, x * a.y - y - a.x); }
  __device__ __host__ vec3 operator*(float a) const { return vec3(x * a, y * a, z * a); }
  __device__ __host__ vec3 operator/(float a) const { return vec3(x / a, y / a, z / a); }
  __device__ __host__ vec3 operator~() const { return (*this) / sqrt(*this * *this); }
  __device__ __host__ float operator+() const { return sqrt(*this * *this); }
  __device__ __host__ vec3 operator&&(const vec3& a) const { return vec3(x * a.x, y * a.y, z * a.z); }
  __device__ __host__ vec3 operator!() const { return vec3(x < 0 ? -x : x, y < 0 ? -y : y, z < 0 ? -z : z); }

  void print(const char* s = "") const { printf("%s %lg %lg %lg\n", s, x, y, z); }
};

struct ray {
  vec3 p, d, w;
  __device__ __host__ ray(const vec3& p = vec3(), const vec3& d = vec3()):p(p),d(d),w(vec3(1, 1, 1)) {}
};

struct plane {
  vec3 p, n;
};

struct sphere {
  vec3 p;
  float r;
};

struct obj {
  enum {PLANE, SPHERE, NOTHING} type;
  union {
    plane p;
    sphere s;
  };
  vec3 col;
  bool emit;
  obj():type(NOTHING){}
};

#ifdef USE_CUDA
__device__
#endif
float intersect_sphere(const sphere& s, const ray& r, vec3* n) {
  float a = r.d * r.d;
  float b = r.d*(r.p - s.p)*2;
  float c = (r.p - s.p)*(r.p - s.p)-s.r*s.r;
  float d = b*b - 4*a*c;
  if (d < 0) return -1;
  float t0 = (-b-sqrt(d))/2/a;
  float t1 = (-b+sqrt(d))/2/a;
  float t;
  if (t0 < 0) t = t1;
  else t = t0;
  vec3 p = r.p + r.d * t;
  *n = ~(p - s.p);
  return t;
}

#ifdef USE_CUDA
__device__
#endif
float intersect_plane(const plane& p, const ray& r, vec3* n) {
  *n = p.n;
  return (p.p - r.p) * p.n / (r.d * p.n);
}

#ifdef USE_CUDA
__device__
#endif
float intersect(const obj& o, const ray& r, vec3* n) {
  switch (o.type) {
    case obj::PLANE: return intersect_plane(o.p, r, n);
    case obj::SPHERE: return intersect_sphere(o.s, r, n);
  }
  return -1;
}

ray rays[600 * 600];
obj objs[8];
ray* rays_;
obj* objs_;
vec3* screen_;

__global__ void setup_kernel(hiprandState *state) {
#ifdef USE_CUDA
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  curand_init(1234, i, 0, &state[i]);
#endif
}

hiprandState *r_state;
__device__ __host__ vec3 rv(hiprandState* state, const vec3& n) {
  float x, y, z, d;
  vec3 t;
#ifdef USE_CUDA
  int i = threadIdx.x + blockDim.x * blockIdx.x;
#endif
  do {
#ifdef USE_CUDA
    x = curand_uniform(&state[i]) * 2 - 1;
    y = curand_uniform(&state[i]) * 2 - 1;
    z = curand_uniform(&state[i]) * 2 - 1;
#else
    x = rand() * 1. / RAND_MAX * 2 - 1;
    y = rand() * 1. / RAND_MAX * 2 - 1;
    z = rand() * 1. / RAND_MAX * 2 - 1;
#endif
    d = sqrt(x*x + y*y + z*z);
    t = vec3(x/d, y/d, z/d);
  } while (d > 1 && n*t < 0);
  return t;
}

#ifdef USE_CUDA
__global__
#endif
void trace(hiprandState *state, ray* rays, obj* objs, vec3* screen, int i_) {
#ifdef USE_CUDA
  int i = threadIdx.x + blockDim.x * blockIdx.x;
#else
  int i = i_;
#endif
  if (i > 600 * 600) return;
  if (+rays[i].w < 0.01) return;
  float t = -1;
  obj* o = NULL;;
  vec3 n;
  for (int j = 0; j < 8; j++) {
    vec3 n_ = vec3(0, 0, 0);
    float t0 = intersect(objs[j], rays[i], &n_);
    if (t0 > 0.001 && t < 0 || t0 > 0.001 && t0 < t) {
      t = t0;
      o = &objs[j];
      n = n_;
    }
  }
    /* if (t < 0) screen[i] = vec3(); */
    /* else screen[i] = !n; //vec3(1/t, 1/t, 1/t); */
    /* return; */
  if (t > 0.001) {
    /* screen[i] = (*o).col; */
    /* return; */
    if (n * rays[i].d > 0) {
      rays[i].w = vec3();
      //screen[i] = vec3(1, 0, 1);
      return;
    } else {
      // screen[i] = (*o).col;
    }
    if ((*o).emit) {
      screen[i] = screen[i] + ((*o).col && rays[i].w);
      rays[i].w = vec3();
      return;
    }
    float diff = n * rays[i].d * -1;
    diff = diff > 0 ? diff : 0;
    rays[i].p = rays[i].p + rays[i].d * t;
    rays[i].d = rv(state, n);
    rays[i].w = (rays[i].w && (*o).col) * diff * 0.7;
  } else {
    rays[i].w = vec3();
  }
}

__device__ __host__ float r(hiprandState* state) {
#ifdef USE_CUDA
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  return curand_uniform(&state[i]) * 2 - 1;
#else
  return 1. * rand() / RAND_MAX * 2 - 1;
#endif
}

#ifdef USE_CUDA
__global__
#endif
void genRays(hiprandState* state, ray* rays, int i_) {
#ifdef USE_CUDA
  int i = threadIdx.x + blockIdx.x * blockDim.x;
#else
  int i = i_;
#endif
  if (i > 600 * 600) return;

  vec3 eye(0, 0, 5);
  vec3 lookat(0, 0, 2);
  vec3 up(0, 1, 0);
  vec3 right(1, 0, 0);
  // up = ~((eye - lookat) % right);
  int x = i % 600;
  int y = i / 600;
  vec3 t = lookat + right * ((x - 300 + r(state)) / 300.) + up * ((y - 300 + r(state)) / 300.);
  rays[i] = ray(eye, ~(t - eye));
}

void err(const char* s = "") {
  // printf("%s: %s\n", s, cudaGetErrorString(cudaGetLastError()));
}

void render(float* s) {
  vec3* screen = (vec3*) s;
  for (int i = 0; i < 360000; i++) {
    screen[i] = screen[i] * 0.99;
  }

#ifdef USE_CUDA
  cudaMemcpy(screen_, screen, 600*600*sizeof(vec3), cudaMemcpyHostToDevice); err("memcpy screen");
#endif

  for (int i = 0; i < 10; i++) {
#ifdef USE_CUDA
    genRays<<<600, 600>>>(r_state, rays_, 0);
#else
    for (int k = 0; k < 360000; k++) genRays(r_state, rays_, k);
#endif

    for (int j = 0; j < 6; j++) {
#ifdef USE_CUDA
      trace<<<600, 600>>>(r_state, rays_, objs_, screen_, 0); err("call trace");
#else
    for (int k = 0; k < 360000; k++) trace(r_state, rays_, objs, screen, k);
#endif
    }
  }

#ifdef USE_CUDA
  cudaMemcpy(screen, screen_, 600*600*sizeof(vec3), cudaMemcpyDeviceToHost); err("back memcpy screen");
#endif

  float ml = 0;
  for (int i = 0; i < 360000; i++) {
    float l = .21 * screen[i].x + 0.71 * screen[i].y + .07 * screen[i].z;
    ml = l > ml ? l : ml;
  }
  ml = ml / (1 + ml);
  for (int i = 0; i < 360000; i++) {
    screen[i] = screen[i];;
    /* screen[i].x = screen[i].x > 1 ? 1 : screen[i].x; */
    /* screen[i].y = screen[i].y > 1 ? 1 : screen[i].y; */
    /* screen[i].z = screen[i].z > 1 ? 1 : screen[i].z; */
  }
}

void init() {
#ifdef USE_CUDA
  cudaMalloc(&r_state, 600*600*sizeof(curandState)); err("malloc rand");
  cudaMalloc(&rays_, 600*600*sizeof(ray)); err("malloc rays");
  cudaMalloc(&objs_, 8*sizeof(obj)); err("malloc objs");
  cudaMalloc(&screen_, 600*600*sizeof(vec3)); err("malloc screen");
#else
  rays_ = (ray*)malloc(600*600*sizeof(ray));
#endif

  setup_kernel<<<600, 600>>>(r_state); err("call rand_setup");
  objs[0].type = obj::SPHERE;
  objs[0].col = vec3(1, 1, 1);
  objs[0].s.p = vec3(0.5, -0.5, 0);
  objs[0].s.r = 0.4;

  objs[1].type = obj::SPHERE;
  objs[1].s.p = vec3(0, 0.85, -0.85);
  objs[1].s.r = 0.1;
  objs[1].emit = true;
  objs[1].col = vec3(0.5, 0.5, 0.5);

  objs[2].type = obj::SPHERE;
  objs[2].col = vec3(1, 1, 1);
  objs[2].s.p = vec3(-0.5, -0.5, 0);
  objs[2].s.r = 0.4;

  objs[3].type = obj::PLANE;
  objs[3].col = vec3(1, 1, 1);
  objs[3].p.p = vec3(0, -1, 0);
  objs[3].p.n = vec3(0, 1, 0);

  objs[4].type = obj::PLANE;
  objs[4].col = vec3(0, 1, 0);
  objs[4].p.p = vec3(-1, 0, 0);
  objs[4].p.n = vec3(1, 0, 0);

  objs[5].type = obj::PLANE;
  objs[5].col = vec3(1, 0, 0);
  objs[5].p.p = vec3(1, 0, 0);
  objs[5].p.n = vec3(-1, 0, 0);

  objs[6].type = obj::PLANE;
  objs[6].col = vec3(1, 1, 1);
  objs[6].p.p = vec3(0, 1, 0);
  objs[6].p.n = vec3(0, -1, 0);

  objs[7].type = obj::PLANE;
  objs[7].col = vec3(1, 1, 1);
  objs[7].p.p = vec3(0, 0, -1);
  objs[7].p.n = vec3(0, 0, 1);

#ifdef USE_CUDA
  cudaMemcpy(objs_, objs, 8*sizeof(obj), cudaMemcpyHostToDevice); err("memcpy objs");
#endif
}
